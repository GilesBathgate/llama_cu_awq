#include "hip/hip_runtime.h"
﻿/*
Inference for Llama-2 Transformer model in pure Cuda.

### INT4 - AWQ quantization version ###

1. First generate AWQ int-4 quantized weights following steps in https://github.com/mit-han-lab/llm-awq
 E.g:
  python -m awq.entry --model_path /path-to-model/Llama-2-7b-chat-hf --w_bit 4 --q_group_size 128 --run_awq --dump_awq awq_cache/llama2-7b-chat-metadata.pt
  python -m awq.entry --model_path /path-to-model/Llama-2-7b-chat-hf --w_bit 4 --q_group_size 128 --load_awq awq_cache/llama2-7b-chat-metadata.pt --q_backend real --dump_quant awq_weights/llama2-7b-awq.pt
 Note - AWQ scripts doesn't run on Windows. Use Linux or WSL.

2. Convert AWQ weights into individual weight binary files using convert_awq_to_bin.py

3. Convert/repack the weight binary files using the weight_packer.cpp utility.

4. Run this program pointing to the final weight file.
*/

#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>

constexpr int group_size = 128; // hardcoded for this implementation
#define DUMP_PER_TOKEN_TIMINGS 0
#define USE_CUDA_GRAPHS 1

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void copy_embedding_kernel(half* x, const half* __restrict__ table, int size, int* tokens, int* pPos)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    int pos = *pPos;
    int token = tokens[pos];
    int table_index = index + token * size;
    x[index] = table[table_index];
}

// Single block - not enough parallelism for the GPU, but it's just 1% of total time
__global__ void rmsnorm_kernel(half* o, half* x, half* weight, int size, int elementsPerThread) {
    float ss = 0.0f;
    for (int i = 0; i < elementsPerThread; i++) {
        int index = threadIdx.x + i * 1024;
        if (index < size) {
            float val = (float)x[index];
            ss += val * val;
        }
    }

    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    ss = BlockReduce(temp).Sum(ss);

    __shared__ float shared_ss;
    if (threadIdx.x == 0) {
        ss /= size;
        ss += 1e-6f;
        ss = 1.0f / sqrtf(ss);
        shared_ss = ss;
    }
    __syncthreads();
    ss = shared_ss;

    // normalize
    for (int i = 0; i < elementsPerThread; i++) {
        int index = threadIdx.x + i * 1024;
        if (index < size) {
            float val = (float)x[index];
            val *= ss * (float)weight[index];
            o[index] = (half)val;
        }
    }
}


// Only used for the final linear layer to get logits (for most other layers we use the INT4 version below)
__global__ void mat_vec_kernel(half* op, const half* ip, const half* wt, int n, int d, int numSerialLoads, 
    int ip_stride, int w_stride, int op_stride, int w_row_stride, float alpha) {
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index >= d)
        return;
    const half* __restrict__ input = ip + blockIdx.y * ip_stride;
    const half* __restrict__ weight = wt + blockIdx.y * w_stride;
    half* output = op + blockIdx.y * op_stride;

    float sum = 0;

    for (int i = 0; i < numSerialLoads; i++) {
        int j = (i * 32 + threadIdx.x) * 8;
        if (j < n) {
            half w[8];
            half ip[8];
            *((uint4 *)(&w)) = *((uint4 *)(&weight[index * w_row_stride + j]));
            *((uint4 *)(&ip)) = *((uint4 *)(&input[j]));
            for (int el = 0; el < 8; el++)
                sum += float(w[el]) * float(ip[el]);
        }
    }

    using WarpReduce = hipcub::WarpReduce<float>;
    __shared__ typename WarpReduce::TempStorage temp;
    sum = WarpReduce(temp).Sum(sum);
    sum *= alpha;

    if (threadIdx.x == 0)
        output[index] = (half)sum;
}

// Simpler version of the above - handles non multiple of 8 dimensions too (used only by MHA block)
__global__ void mat_vec_kernel_simple(half* op, half* ip, half* wt, int n, int numSerialElements,
    int ip_stride, int w_stride, int w_row_stride, float alpha, int *pPos) {

    int op_stride = *pPos + 1;
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index >= op_stride)
        return;

    const half* __restrict__ input = ip + blockIdx.y * ip_stride;
    const half* __restrict__ weight = wt + blockIdx.y * w_stride;
    half* output = op + blockIdx.y * op_stride;

    float sum = 0;
    for (int i = 0; i < numSerialElements; i++) {
        int j = i * 32 + threadIdx.x;
        if (j < n)
            sum += ((float)weight[index * w_row_stride + j]) * ((float)input[j]);
    }

    using WarpReduce = hipcub::WarpReduce<float>;
    __shared__ typename WarpReduce::TempStorage temp;
    sum = WarpReduce(temp).Sum(sum);
    sum *= alpha;

    if (threadIdx.x == 0)
        output[index] = (half)sum;
}

// hardcoded for group-count = 128
__global__ void mat_vec_kernel_int4(half* __restrict__ output, const half* __restrict__ input,
    const uint32_t* __restrict__ q_weight, const uint32_t* __restrict__ q_zeros, const half* __restrict__ scales,
    int inputElements, int opElements, int packed_zeros_height, int scales_height, int packed_weights_height, bool accum, int loff, int *pPos)
{
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index >= opElements)
        return;

    float sum = 0;
    for (int ygq = 0; ygq * 128 + threadIdx.x * 4 < packed_weights_height; ygq++) {   // each iteration of this loop covers 8 x 128 elements in y dimension of weight matrix (weight matrix is column major)
        uint32_t packed_q_z = q_zeros[index * packed_zeros_height + ygq];

        // load weights in one go (32 elements from weight matrix loaded by each thread in one read)
        uint32_t loaded_packed_wts[4];
        *((uint4*)(&loaded_packed_wts[0])) = *((uint4*)(&q_weight[index * packed_weights_height + ygq * 128 + threadIdx.x * 4]));

        int group_y = ygq * 8 + (threadIdx.x / 4);
        float q_z = (float)(packed_q_z >> (4 * (threadIdx.x / 4)) & 0xF);
        float scale = (float)scales[index * scales_height + group_y];
        int y_base = ygq * 1024 + threadIdx.x * 32;

        for (int qi = 0; qi < 4; qi ++) {                 // each iteration of this loop covers 256 elements in y dimension of weight matrix
            int ys = y_base + qi * 8;
            if (ys < inputElements) {
                uint32_t packed_q_w = loaded_packed_wts[qi];
                half ip[8];
                *((uint4*)(&ip)) = *((uint4*)(&input[ys]));

                for (int i = 0; i < 8; i++) {
                    float q_wt = (float)(packed_q_w & 0xF);
                    float w = (q_wt - q_z) * scale;
                    sum += w * float(ip[i]);
                    packed_q_w = (packed_q_w >> 4);
                }
            }
        }
    }

    using WarpReduce = hipcub::WarpReduce<float>;
    __shared__ typename WarpReduce::TempStorage temp;
    sum = WarpReduce(temp).Sum(sum);

    if (threadIdx.x == 0) {
        if (loff != -1) {
            output += loff + (*pPos * opElements);
        }

        if (accum)
            sum += (float)output[index];
        output[index] = (half)sum;
    }
}

// Here we make use of shared memory to achieve better memory access pattern, and transpose a 32x32 chunk of the matrix on the fly
// Again used only by the MHA block
__global__ void vec_mat_kernel(half* op, const half* __restrict__ ip, const half* __restrict__ wt, int N, int *pPos, int w_stride, int op_stride, int w_row_stride) {
    int K = *pPos + 1;
    const half* __restrict__ input = ip + blockIdx.y * K;
    const half* __restrict__ weight = wt + blockIdx.y * w_stride;
    half* output = op + blockIdx.y * op_stride;

    int start_n = blockIdx.x * 32;
    int i = start_n + threadIdx.y;

    // 2x for double buffering
    // +2 to avoid shared memory bank conflicts
    __shared__ half loaded_fragment[2][32][32 + 2];

    // OOB check
    if (i >= N)
        return;

    // load the first 32x32 fragment
    int n = start_n + threadIdx.x;
    int k = threadIdx.y;
    int offset = k * w_row_stride + n;
    loaded_fragment[0][threadIdx.y][threadIdx.x] = ((n < N) && (k < K)) ? weight[offset] : 0;

    float sum = 0;
    // Loop over the matrix row and vector elements
    for (int e = 0; ;) {
        __syncthreads();    // wait for the load

        int start_k = e * 32;
        if (start_k >= K) break;
        k = start_k + threadIdx.x;
        int buf_i = e & 1;
        sum += float(loaded_fragment[buf_i][threadIdx.x][threadIdx.y]) * ((k < K) ? (float) input[k] : 0);

        // load for the next iteration
        e++;
        start_k = e * 32;
        buf_i = e & 1;
        n = start_n + threadIdx.x;
        k = start_k + threadIdx.y;
        int offset = k * w_row_stride + n;
        loaded_fragment[buf_i][threadIdx.y][threadIdx.x] = ((n < N) && (k < K)) ? weight[offset] : 0;
    }

    using WarpReduce = hipcub::WarpReduce<float>;
    __shared__ typename WarpReduce::TempStorage temp;
    sum = WarpReduce(temp).Sum(sum);

    if (threadIdx.x == 0)
        output[i] = (half)sum;
}

// Each block processes a single head
__global__ void RoPERotation_kernel(half* sq, half* sk_base, int num_heads, int head_size, int *pPos, int loff) {
    int pos = *pPos;
    half* sk = sk_base + loff + pos * num_heads * head_size;
    int h = blockIdx.x;
    half* q = sq + h * head_size;
    half* k = sk + h * head_size;
    int i = threadIdx.x;
    int head_dim = (i * 2) % head_size;
    float freq = 1.0f / powf(10000.0f, head_dim / (float)head_size);
    float val = pos * freq;
    float fcr = cosf(val);
    float fci = sinf(val);
    float q0 = q[i];
    float q1 = q[i + head_size/2];
    float k0 = k[i];
    float k1 = k[i + head_size / 2];
    q[i] = q0 * fcr - q1 * fci;
    q[i + head_size / 2] = q0 * fci + q1 * fcr;
    k[i] = k0 * fcr - k1 * fci;
    k[i + head_size / 2] = k0 * fci + k1 * fcr;
}

#define MAX_SEQ_LEN 8192
__global__ void softmax_kernel(half* __restrict__ arr, int num_heads, int *pPos) {
    __shared__ float att[MAX_SEQ_LEN];
    int h = blockIdx.x;
    int tid = threadIdx.x;
    int step = blockDim.x;
    int size = *pPos + 1;

    // load input to shared memory
    for (int t = tid; t < size; t += step)
        att[t] = (float) arr[h * size + t];
    __syncthreads();

    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;

    // find max value (for numerical stability)
    float max_val = tid < size ? att[tid] : 0;
    for (int i = tid + step; i < size; i += step)
        if (att[i] > max_val)
            max_val = att[i];

    max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0)
        shared_val = max_val;
    __syncthreads();
    max_val = shared_val;

    // exp and sum
    float sum = 0.0f;
    for (int i = tid; i < size; i += step) {
        att[i] = expf(att[i] - max_val);
        sum += att[i];
    }

    sum = BlockReduce(temp).Sum(sum);
    if (threadIdx.x == 0)
        shared_val = sum;
    __syncthreads();
    sum = shared_val;

    // normalize and write the result
    for (int t = tid; t < size; t += step)
        arr[h * size + t] = (half) (att[t] / sum);
}

__global__ void silu_element_wise_mul_kernel(half* dest, half* src, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float val = (float)dest[i];
        val *= 1.0f / (1.0f + expf(-val));
        val *= (float)src[i];
        dest[i] = (half)val;
    }
}

__global__ void argmax_kernel(half* __restrict__ x, int size, int* result, volatile int* pPos, int* pPosGpu, bool write_token) {
    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;

    int tid = threadIdx.x;
    int step = blockDim.x;

    // find local max value and its position
    float max_val = tid < size ? (float)x[tid] : -INFINITY;
    int   max_pos = tid < size ? tid : 0;
    for (int i = tid + step; i < size; i += step) {
        if ((float)x[i] > max_val) {
            max_val = x[i];
            max_pos = i;
        }
    }

    // find the global max value
    float global_max_val;
    global_max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0)
        shared_val = global_max_val;
    __syncthreads();
    global_max_val = shared_val;

    // possibility of race condition here, so we first write it to shared memory variable and then have just one thread to update the pointers.
    __shared__ int global_max_pos;
    if (max_val == global_max_val) {
        global_max_pos = max_pos;
    }
    __syncthreads();

    // write next token to the current token location
    if (threadIdx.x == 0) {
        int token_pos = *pPos;
        token_pos++;

        if (write_token)
            result[token_pos] = global_max_pos;

        // update the token indices (unblocks the CPU)
        *pPos = token_pos;
        *pPosGpu = token_pos;
    }
}

// ----------------------------------------------------------------------------
// Transformer and RunState structs, and related memory management

typedef struct {
    int dim; // transformer dimension
    int hidden_dim; // for ffn layers
    int n_layers; // number of layers
    int n_heads; // number of query heads
    int n_kv_heads; // number of key/value heads (can be < query heads because of multiquery)
    int vocab_size; // vocabulary size, usually 256 (byte-level)
    int seq_len; // max sequence length
} Config;

struct QWeight {
    uint32_t* weight;
    uint32_t* zeros;
    half* scales;
};

struct PerLayerWeight {
    half* rms_att_weight; // (layer, dim) rmsnorm weights
    half* rms_ffn_weight; // (layer, dim)
    QWeight wq_q;
    QWeight wq_k;
    QWeight wq_v;
    QWeight wq_o;
    QWeight wq_gate;
    QWeight wq_up;
    QWeight wq_down;
};

typedef struct {
    // token embedding table
    half* token_embedding_table;    // (vocab_size, dim)
    // classifier weights for the logits, on the last layer
    half* wcls;
    // final rmsnorm
    half* rms_final_weight; // (dim,)
    // Per layer weights
    PerLayerWeight* layers;
    int num_layers;
} TransformerWeights;

// data shared between CPU and GPU (allocated in host memory)
struct SharedData {
    volatile int pos;         // current token index
    int tokens[MAX_SEQ_LEN];  // seq_len (tokens processed/generated so far) allocated in host memory so that CPU can read this
};

typedef struct {
    // current wave of activations
    half* x; // activation at current time stamp (dim,)
    half* xb; // same, but inside a residual branch (dim,)
    half* hb; // buffer for hidden dimension in the ffn (hidden_dim,)
    half* hb2; // buffer for hidden dimension in the ffn (hidden_dim,)
    half* q; // query (dim,)
    half* att; // buffer for scores/attention values (n_heads, seq_len)
    half* logits; // output logits
    // kv cache
    half* key_cache;   // (layer, seq_len, dim)
    half* value_cache; // (layer, seq_len, dim)

    int* pos;  // GPU copy of the current position (just 1 element)
    SharedData* shared_data;
} RunState;

void malloc_run_state(RunState* s, Config* p) {
    hipMalloc((void**)&s->x, p->dim * sizeof(half));
    hipMalloc((void**)&s->xb, p->dim * sizeof(half));
    hipMalloc((void**)&s->hb, p->hidden_dim * sizeof(half));
    hipMalloc((void**)&s->hb2, p->hidden_dim * sizeof(half));
    hipMalloc((void**)&s->q, p->dim * sizeof(half));
    hipMalloc((void**)&s->att, p->n_heads * p->dim * sizeof(half));
    hipMalloc((void**)&s->logits, p->vocab_size * sizeof(half));
    hipMalloc((void**)&s->key_cache, p->n_layers * p->seq_len * p->dim * sizeof(half));    // potentially huge allocs
    hipMalloc((void**)&s->value_cache, p->n_layers * p->seq_len * p->dim * sizeof(half));

    hipMalloc((void**)&s->pos, sizeof(int));
    hipHostMalloc((void**)&s->shared_data, sizeof(SharedData));

    // ensure all mallocs went fine
    if (!s->x || !s->xb || !s->pos || !s->hb || !s->hb2 || !s->q
        || !s->att || !s->logits || !s->key_cache
        || !s->value_cache || !s->shared_data) {
        printf("malloc failed for allocaing run state!\n");
        exit(1);
    }
}

void free_run_state(RunState* s) {
    hipFree(s->x);
    hipFree(s->xb);
    hipFree(s->pos);
    hipFree(s->hb);
    hipFree(s->hb2);
    hipFree(s->q);
    hipFree(s->att);
    hipFree(s->logits);
    hipFree(s->key_cache);
    hipFree(s->value_cache);
    hipHostFree(s->shared_data);
}

int divUp(int a, int b) {
    return (a - 1) / b + 1;
}

size_t getPackedWeightHeight(size_t height)
{
    // Each uint32 element in the packed weight matrix contain 8 elements from the original matrix.
    // Also we load 4 uint's (32 elements) in a single instruction for getting better memory efficiency
    // This requires us to align the "height" dimension to a multiple of 4 uint (or 32 elements)
    return divUp(height, 32) * 4;
}

void allocQWeight(QWeight* pWeight, size_t height, size_t width) {
    size_t packed_wt_height = getPackedWeightHeight(height);
    size_t scales_height = divUp(height, group_size);
    size_t packed_zeros_height = divUp(scales_height, 8);

    hipMalloc((void**)&pWeight->weight, packed_wt_height * width * sizeof(uint32_t));
    hipMalloc((void**)&pWeight->zeros, packed_zeros_height * width * sizeof(uint32_t));
    hipMalloc((void**)&pWeight->scales, scales_height * width * sizeof(half));
}

void freeQWeight(QWeight* pWeight) {
    hipFree(pWeight->weight);
    hipFree(pWeight->zeros);
    hipFree(pWeight->scales);
}

void malloc_weights(TransformerWeights* w, Config* p) {
    hipMalloc((void**)&w->token_embedding_table, p->vocab_size * p->dim * sizeof(half));
    w->layers = (PerLayerWeight*)malloc(p->n_layers * sizeof(PerLayerWeight));
    w->num_layers = p->n_layers;
    for (int l = 0; l < p->n_layers; l++)
    {
        PerLayerWeight* layer = &(w->layers[l]);
        hipMalloc((void**)&layer->rms_att_weight,  p->dim * sizeof(half));
        hipMalloc((void**)&layer->rms_ffn_weight,  p->dim * sizeof(half));
        allocQWeight(&layer->wq_q, p->dim, p->dim);
        allocQWeight(&layer->wq_k, p->dim, p->dim);
        allocQWeight(&layer->wq_v, p->dim, p->dim);
        allocQWeight(&layer->wq_o, p->dim, p->dim);
        allocQWeight(&layer->wq_gate, p->dim, p->hidden_dim);
        allocQWeight(&layer->wq_up, p->dim, p->hidden_dim);
        allocQWeight(&layer->wq_down, p->hidden_dim, p->dim);
    }

    hipMalloc((void**)&w->rms_final_weight, p->dim * sizeof(half));
    int head_size = p->dim / p->n_heads;
    hipMalloc((void**)&w->wcls, p->vocab_size * p->dim * sizeof(half));

    // ensure all mallocs went fine
    if (!w->token_embedding_table || !w->layers ||
        !w->rms_final_weight || !w->wcls) {
        printf("malloc failed!\n");
        exit(1);
    }
}

void free_weights(TransformerWeights* w) {
    hipFree(w->token_embedding_table);
    hipFree(w->rms_final_weight);
    hipFree(w->wcls);
    for (int l = 0; l < w->num_layers; l++) {
        PerLayerWeight* layer = &(w->layers[l]);
        hipFree(layer->rms_att_weight);
        hipFree(layer->rms_ffn_weight);
        freeQWeight(&layer->wq_q);
        freeQWeight(&layer->wq_k);
        freeQWeight(&layer->wq_v);
        freeQWeight(&layer->wq_o);
        freeQWeight(&layer->wq_gate);
        freeQWeight(&layer->wq_up);
        freeQWeight(&layer->wq_down);
    }
    free(w->layers);
}

// ----------------------------------------------------------------------------
// initialization: read from checkpoint
void readWeight(void* op, FILE* fp, size_t bytes, void* scratch) {
    if (fread(scratch, 1, bytes, fp) != bytes) { printf("error reading weights");  exit(1); }
    hipMemcpyAsync(op, scratch, bytes, hipMemcpyHostToDevice);
}

void uploadQWeight(QWeight& weight, FILE* fp, size_t height, size_t width, void* scratch) {
    int meta_height = divUp(height, group_size);
    int packed_wt_height = getPackedWeightHeight(height);
    int packed_zeros_height = divUp(meta_height, 8);

    readWeight(weight.weight, fp, packed_wt_height * width * sizeof(uint32_t), scratch);
    readWeight(weight.zeros,  fp, packed_zeros_height * width * sizeof(uint32_t), scratch);
    readWeight(weight.scales, fp, meta_height * width * sizeof(half), scratch);
}

int checkpoint_init_weights(TransformerWeights* w, Config* p, FILE* f) {
    size_t scratch_size = std::max(p->vocab_size, p->hidden_dim) * p->dim;
    scratch_size *= sizeof(half);
    void* scratchCpu = malloc(scratch_size);

    readWeight(w->token_embedding_table, f, p->vocab_size * p->dim * sizeof(half), scratchCpu);
    readWeight(w->wcls, f, p->vocab_size * p->dim * sizeof(half), scratchCpu);
    readWeight(w->rms_final_weight, f, p->dim * sizeof(half), scratchCpu);

    // upload decoder block weight for each layer
    for (int i = 0; i < p->n_layers; i++) {
        uploadQWeight(w->layers[i].wq_q, f, p->dim, p->dim, scratchCpu);
        uploadQWeight(w->layers[i].wq_k, f, p->dim, p->dim, scratchCpu);
        uploadQWeight(w->layers[i].wq_v, f, p->dim, p->dim, scratchCpu);
        uploadQWeight(w->layers[i].wq_o, f, p->dim, p->dim, scratchCpu);

        uploadQWeight(w->layers[i].wq_up  , f, p->dim, p->hidden_dim, scratchCpu);
        uploadQWeight(w->layers[i].wq_gate, f, p->dim, p->hidden_dim, scratchCpu);
        uploadQWeight(w->layers[i].wq_down, f, p->hidden_dim, p->dim, scratchCpu);

        readWeight(w->layers[i].rms_att_weight, f, p->dim * sizeof(half), scratchCpu);
        readWeight(w->layers[i].rms_ffn_weight, f, p->dim * sizeof(half), scratchCpu);
    }

    printf("\nloaded weights\n");
    free(scratchCpu);
    return 0;
}


// ----------------------------------------------------------------------------
// neural net blocks
hipStream_t stream;

void rmsnorm(half* o, half* x, half* weight, int size) {
    int elementsPerThread = divUp(size, 1024);
    rmsnorm_kernel <<< 1, 1024, 0, stream>>> (o, x, weight, size, elementsPerThread);
}

void matmul(half* xout, half* x, half* w, int n, int d, int batch = 1, int x_stride = 0, int w_stride = 0, int op_stride = 0, int w_row_stride = -1, float alpha = 1.0f) {
    if ((n & 7) || (d & 7)) { printf("\nUnsupported matmul size. Exiting\n"); exit(1); }
    int serialElements = divUp(n, 32);
    int serialLoads = divUp(serialElements, 8);     // we load 8 elements in parallel
    dim3 block_dim(32, 4);
    dim3 grid_dim(divUp(d, 4), batch);
    if (w_row_stride == -1) w_row_stride = n;
    mat_vec_kernel <<<grid_dim, block_dim, 0, stream >>> (xout, x, w, n, d, serialLoads, x_stride, w_stride, op_stride, w_row_stride, alpha);
}

void matmul(half* xout, half* x, QWeight &w, int inpSize, int opSize, bool accum = false, int loff = -1, int *pPos = nullptr) {
    if ((inpSize & 7) || (opSize & 7)) { printf("\nUnsupported matmul size. Exiting\n"); exit(1); }
    // We are assuming a vector - matrix mul with col major matrix: height = inpSize,  width  = opSize
    int scales_height = divUp(inpSize, 128);
    int packed_wt_height = getPackedWeightHeight(inpSize);
    int packed_zeros_height = divUp(scales_height, 8);
    dim3 block_dim(32, 4);
    dim3 grid_dim(divUp(opSize, 4), 1);
    mat_vec_kernel_int4 <<<grid_dim, block_dim, 0, stream >>> (xout, x, w.weight, w.zeros, w.scales, inpSize, opSize, packed_zeros_height, scales_height, packed_wt_height, accum, loff, pPos);
}

void RoPERotation(half *q, half *k, int num_heads, int head_size, int* pPos, int loff) {
    RoPERotation_kernel <<<num_heads, head_size / 2, 0, stream >>> (q, k, num_heads, head_size, pPos, loff);
}

void MultiHeadAttention(half *output, half *q, half *key_cache, half * value_cache, half *att, int num_heads, int head_size, int max_seq_len, int *pPos) {
    int dim = head_size * num_heads;
    // 1. Get attention scores
    int serialElements = divUp(head_size, 32);
    dim3 block_dim(32, 32);
    dim3 grid_dim1(divUp(max_seq_len, 32), num_heads);      // using max_seq_len instead of real seq_len here has measurable impact on perf (2%) :-/
    mat_vec_kernel_simple <<< grid_dim1, block_dim, 0, stream >>> (att, q, key_cache, head_size, serialElements, head_size, head_size, dim, 1.0 / sqrt(head_size), pPos);

    // 2. Run softmax kernel
    softmax_kernel <<< num_heads, 1024, 0, stream >>> (att, num_heads, pPos);

    // 3. weighted sum of the values to get the final result
    dim3 grid_dim2(divUp(head_size, 32), num_heads);
    vec_mat_kernel <<< grid_dim2, block_dim, 0, stream >>> (output, att, value_cache, head_size, pPos, head_size, head_size, dim);
}

void siluElementwiseMul(half *hb, half *hb2, int size) {
   silu_element_wise_mul_kernel <<< divUp(size, 256), 256, 0, stream >>> (hb, hb2, size);
}

void run_llama_network(int *pPos, Config* p, RunState* s, TransformerWeights* w, int seq_len_bin) {
    half* x = s->x;
    int dim = p->dim;
    int hidden_dim = p->hidden_dim;
    int head_size = dim / p->n_heads;

    copy_embedding_kernel <<<divUp(dim, 256), 256, 0, stream >>> (x, w->token_embedding_table, dim, s->shared_data->tokens, pPos);
    
    // forward all the layers
    for (int l = 0; l < p->n_layers; l++) {

        // attention rmsnorm
        rmsnorm(s->xb, x, w->layers[l].rms_att_weight, dim);

        // we directly store (key, value) at this time step (pos) to our kv cache
        int loff = l * p->seq_len * dim; // kv cache layer offset for convenience

        // qkv matmuls for this position (opt: can be done in single kernel as batch of 3)
        matmul(s->q, s->xb, w->layers[l].wq_q, dim, dim);
        matmul(s->key_cache, s->xb, w->layers[l].wq_k, dim, dim, false, loff, pPos);
        matmul(s->value_cache, s->xb, w->layers[l].wq_v, dim, dim, false, loff, pPos);

        // apply RoPE rotation to the q and k vectors for each head
        // also save the output (key, value) at this time step (pos) to our kv cache
        RoPERotation(s->q, s->key_cache, p->n_heads, head_size, pPos, loff);

        // apply MHA using the query and the key-value cache
        MultiHeadAttention(s->xb, s->q, s->key_cache + loff, s->value_cache + loff, s->att, p->n_heads, head_size, seq_len_bin, pPos);

        // final matmul to get the output of the attention fused with residual connection back into x
        matmul(s->x, s->xb, w->layers[l].wq_o, dim, dim, true);

        // ffn rmsnorm
        rmsnorm(s->xb, x, w->layers[l].rms_ffn_weight, dim);

        // apply gate and up proj (opt: can be done in single kernel as batch of 2)
        matmul(s->hb, s->xb, w->layers[l].wq_gate, dim, hidden_dim);
        matmul(s->hb2, s->xb, w->layers[l].wq_up, dim, hidden_dim);

        // apply F.silu activation on hb and multiply it with hb2
        siluElementwiseMul(s->hb, s->hb2, hidden_dim);
        matmul(s->x, s->hb, w->layers[l].wq_down, hidden_dim, dim, true);
    }

    // final rmsnorm
    rmsnorm(x, x, w->rms_final_weight, dim);

    // classifier into logits
    matmul(s->logits, x, w->wcls, p->dim, p->vocab_size);
}

#define MAX_GRAPHS 8
hipGraphExec_t cudaGraphInstance[MAX_GRAPHS];
bool graphCaptured[MAX_GRAPHS];

void transformer(bool gen_token, Config* p, RunState* s, TransformerWeights* w) {
#if DUMP_PER_TOKEN_TIMINGS == 1
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);
#endif

    int seq_len = s->shared_data->pos + 1;
#if USE_CUDA_GRAPHS
    int graphIndex;
    int seq_len_bin = 128;
    for (graphIndex = 0; graphIndex < MAX_GRAPHS - 1; seq_len_bin *= 2, graphIndex++)
        if (seq_len <= seq_len_bin) break;
    if ((seq_len > seq_len_bin) || (graphIndex == MAX_GRAPHS - 1)) seq_len_bin = p->seq_len;    // last bin holds max seq len

    if (!graphCaptured[graphIndex])
    {
        hipGraph_t graph = {};
        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
        run_llama_network(s->pos, p, s, w, seq_len_bin);
        hipStreamEndCapture(stream, &graph);
        hipGraphInstantiate(&cudaGraphInstance[graphIndex], graph, 0);
        hipGraphDestroy(graph);
        graphCaptured[graphIndex] = true;
    }
    hipGraphLaunch(cudaGraphInstance[graphIndex], stream);
#else
    run_llama_network(s->pos, p, s, w, seq_len);
#endif

    // sample the next token using greedy argmax sampling: take the token with the highest probability (not included in the graph because of gen_token variable)
    argmax_kernel <<<1, 1024, 0, stream>>> (s->logits, p->vocab_size, &(s->shared_data->tokens[0]), &(s->shared_data->pos), s->pos, gen_token);

#if DUMP_PER_TOKEN_TIMINGS == 1
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf(" t: %g ", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif
}

// ----------------------------------------------------------------------------
// byte pair encoding (BPE) tokenizer, encodes strings into tokens so we can prompt

int str_lookup(char *str, char **vocab, int vocab_size) {
    // find the first perfect match for str in vocab, return its index or -1 if not found
    for (int i = 0; i < vocab_size; i++) {
        if (strcmp(str, vocab[i]) == 0) {
            return i;
        }
    }
    return -1;
}

void bpe_encode(char *text, char **vocab, float *vocab_scores, int vocab_size, unsigned int max_token_length, int *tokens, int *n_tokens) {
    
    // a temporary buffer to merge two consecutive tokens
    char* str_buffer = (char*) malloc((max_token_length*2+1) * sizeof(char)); // *2 for concat, +1 for null terminator

    // first encode every individual byte in the input string
    *n_tokens = 0; // the number of tokens
    for (char *c = text; *c != '\0'; c++) {
        sprintf(str_buffer, "%c", *c);
        int id = str_lookup(str_buffer, vocab, vocab_size);
        if (id == -1) { printf("not good\n"); exit(1);}
        tokens[*n_tokens] = id;
        (*n_tokens)++;
    }

    // merge the best consecutive pair each iteration, according the scores in vocab_scores
    while (1) {
        float best_score = -1e10;
        int best_id = -1;
        int best_idx = -1;

        for (int i=0; i < (*n_tokens-1); i++) {
            // check if we can merge the pair (tokens[i], tokens[i+1])
            sprintf(str_buffer, "%s%s", vocab[tokens[i]], vocab[tokens[i+1]]);
            int id = str_lookup(str_buffer, vocab, vocab_size);
            if (id != -1 && vocab_scores[id] > best_score) {
                // this merge pair exists in vocab! record its score and position
                best_score = vocab_scores[id];
                best_id = id;
                best_idx = i;
            }
        }

        if (best_idx == -1) {
            break; // we couldn't find any more pairs to merge, so we're done
        }

        // merge the consecutive pair (best_idx, best_idx+1) into new token best_id
        tokens[best_idx] = best_id;
        // delete token at position best_idx+1, shift the entire sequence back 1
        for (int i = best_idx+1; i < (*n_tokens-1); i++) {
            tokens[i] = tokens[i+1];
        }
        (*n_tokens)--; // token length decreased
    }

    free(str_buffer);
}

// ----------------------------------------------------------------------------
// utilities

long time_in_ms() {
    // return time in milliseconds, for benchmarking the model speed
    struct timespec time;
    timespec_get(&time, TIME_UTC);
    return time.tv_sec * 1000 + time.tv_nsec / 1000000;
}

// ----------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    // poor man's C argparse
    char *checkpoint = NULL;  // e.g. out/model.bin
    int steps = 256;          // max number of steps to run for, 0: use seq_len
    char *prompt = NULL;      // prompt string

    // 'checkpoint' is necessary arg
    if (argc < 2) {
        printf("Usage: %s <checkpoint_file> [steps] [prompt]\n", argv[0]);
        return 1;
    }
    if (argc >= 2) {
        checkpoint = argv[1];
    }
    if (argc >= 3) {
        steps = atoi(argv[2]);
    }
    if (argc >= 4) {
        prompt = argv[3];
    }

    // read in the model.bin file
    Config config = {};
    TransformerWeights weights;
    {
        FILE* file = nullptr;

        file = fopen(checkpoint, "rb");
        if (!file) { printf("Couldn't open file %s\n", checkpoint); return 1; }
        // read in the config header
        if (fread(&config, sizeof(Config), 1, file) != 1) { return 1; }

        // Dump model config
        printf("\nModel params:- \ndim: %d \nhidden_dim: %d\nn_heads: %d\nn_kv_heads: %d\nn_layers: %d\nseq_len: %d\nvocab_size: %d\n\n",
            config.dim, config.hidden_dim, config.n_heads, config.n_kv_heads, config.n_layers, config.seq_len, config.vocab_size);

        config.vocab_size = abs(config.vocab_size);
        // read in the Transformer weights
        malloc_weights(&weights, &config);
        if (checkpoint_init_weights(&weights, &config, file)) { return 1; }
    }
    // right now we cannot run for more than config.seq_len steps
    if (steps <= 0 || steps > config.seq_len) { steps = config.seq_len; }

    // read in the tokenizer.bin file
    char** vocab = (char**)malloc(config.vocab_size * sizeof(char*));
    float* vocab_scores = (float*)malloc(config.vocab_size * sizeof(float));
    unsigned int max_token_length;
    {
        FILE *file = fopen("tokenizer.bin", "rb");
        if (!file) { printf("couldn't load tokenizer.bin\n"); return 1; }
        if (fread(&max_token_length, sizeof(int), 1, file) != 1) { printf("failed read\n"); return 1; }
        int len;
        for (int i = 0; i < config.vocab_size; i++) {
            if (fread(vocab_scores + i, sizeof(float), 1, file) != 1) { printf("failed read\n"); return 1;}
            if (fread(&len, sizeof(int), 1, file) != 1) { printf("failed read\n"); return 1; }
            vocab[i] = (char *)malloc(len + 1);
            if (fread(vocab[i], len, 1, file) != 1) { printf("failed read\n"); return 1; }
            vocab[i][len] = '\0'; // add the string terminating token
        }
        fclose(file);
    }

    // create and init the application RunState
    RunState state;
    malloc_run_state(&state, &config);
    hipStreamCreate(&stream);

    // process the prompt, if any
    int *prompt_tokens = NULL;
    int num_prompt_tokens = 0;
    prompt_tokens = (int*)malloc(config.seq_len * sizeof(int));

    char input_message[2048];
    strcpy(input_message, prompt);

    while (1)
    {
        if (input_message != NULL) {
            bpe_encode(input_message, vocab, vocab_scores, config.vocab_size, max_token_length, prompt_tokens, &num_prompt_tokens);
        }


        // start the main loop
        long start = time_in_ms();  // used to time our code
        int next;                   // will store the next token in the sequence
        int token = 1;              // init with token 1 (=BOS), as done in Llama-2 sentencepiece tokenizer
        int pos = 0;                // position in the sequence

        // copy the prompt tokens into shared list of tokens (so that GPU can access them).
        // init state
        hipMemset(state.pos, 0, sizeof(int));
        state.shared_data->pos = 0;
        state.shared_data->tokens[0] = token;   // BOS
        memcpy(&state.shared_data->tokens[1], prompt_tokens, sizeof(int) * num_prompt_tokens);

        printf("<s>\n"); // explicit print the initial BOS token for stylistic symmetry reasons
        while (pos < steps) {
            // wait for GPU work for previous iteration to complete
            // the idea is to keep GPU working in parallel with any CPU work (e.g, printing tokens to console).
            hipStreamSynchronize(stream);
            // Perf note: don't put CPU work here "before" calling transformer as it won't overlap with GPU execution.
            transformer(pos >= num_prompt_tokens, &config, &state, &weights); // forward the transformer to get next token

            if (pos > 0)
            {
                next = state.shared_data->tokens[pos];  // Note: this is output token from previous iteration

                // following BOS token (1), sentencepiece decoder strips any leading whitespace (see PR #89)
                char* token_str = (token == 1 && vocab[next][0] == ' ') ? vocab[next] + 1 : vocab[next];
                printf("%s", token_str);
                //printf(" [%d - %s] ", next, token_str);
                fflush(stdout);

                if (next == 2) break; // break if EOS token is reached

                // advance forward
                token = next;
            }
            pos++;
        }

        // report achieved tok/s
        long end = time_in_ms();
        double time = (end - start) / 1000.0;
        int timed_tokens = pos - 1;
        printf("\nachieved tok/s: %f. Tokens: %d, seconds: %g\n", timed_tokens / time, timed_tokens, time);

        printf("enter next prompt: ");
        gets_s(input_message);
    }

    // memory cleanup
    free_run_state(&state);
    free_weights(&weights);
#if USE_CUDA_GRAPHS
    for (int i = 0; i < MAX_GRAPHS; i++)
        if (graphCaptured[i]) hipGraphExecDestroy(cudaGraphInstance[i]);
#endif

    for (int i = 0; i < config.vocab_size; i++) { free(vocab[i]); }
    free(vocab);
    free(vocab_scores);
    if (prompt_tokens != NULL) free(prompt_tokens);
    return 0;
}